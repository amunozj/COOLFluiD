#include "hip/hip_runtime.h"
#include "FiniteVolumeCUDA/FVMCC_ComputeSourceRHSCell.hh"
#include "Framework/MeshData.hh"
#include "Framework/CellConn.hh"
#include "Config/ConfigOptionPtr.hh"
#include "Framework/CudaDeviceManager.hh"
#include "Common/SelfRegistPtr.hh"
#include "Common/CUDA/CFVec.hh"
#include "Framework/CudaTimer.hh"
#include "FiniteVolume/FluxData.hh"
#include "FiniteVolume/KernelData.hh"
#include "FiniteVolume/CellData.hh"

#include "FiniteVolumeCUDA/FiniteVolumeCUDA.hh"
#include "Framework/MethodCommandProvider.hh"
#include "Framework/VarSetListT.hh"
#include "FiniteVolume/LaxFriedFlux.hh"
#include "FiniteVolume/LeastSquareP1PolyRec2D.hh"
#include "FiniteVolume/LeastSquareP1PolyRec3D.hh"
#include "FiniteVolume/BarthJesp.hh"
#include "MHD/MHD2DProjectionConsT.hh"
#include "MHD/MHD3DProjectionConsT.hh"
#include "MHD/MHD2DProjectionPrimT.hh"
#include "MHD/MHD3DProjectionPrimT.hh"
#include "MHD/MHDProjectionPrimToConsT.hh"
#include "FiniteVolumeMHD/LaxFriedFluxTanaka.hh"
#include "MHD/MHD2DProjectionVarSet.hh"   
#include "MHD/MHD3DProjectionVarSet.hh"

#include "Maxwell/Maxwell2DProjectionVarSet.hh"
#include "Maxwell/Maxwell2DProjectionConsT.hh"
#include "FiniteVolumeMaxwell/StegerWarmingMaxwellProjection2D.hh"

#include "MultiFluidMHD/MultiFluidMHDVarSet.hh"
#include "MultiFluidMHD/EulerMFMHD2DHalfConsT.hh"
#include "MultiFluidMHD/EulerMFMHD2DHalfRhoiViTiT.hh"
#include "MultiFluidMHD/EulerMFMHD2DHalfRhoiViTiToConsT.hh"
#include "MultiFluidMHD/EulerMFMHD2DHalfConsToRhoiViTiT.hh"
#include "MultiFluidMHD/EulerMFMHD2DConsT.hh"
#include "MultiFluidMHD/EulerMFMHD2DRhoiViTiT.hh"
#include "MultiFluidMHD/EulerMFMHD2DRhoiViTiToConsT.hh"
#include "MultiFluidMHD/EulerMFMHD2DConsToRhoiViTiT.hh"
#include "FiniteVolumeMultiFluidMHD/AUSMPlusUpFluxMultiFluid.hh"
#include "FiniteVolumeMultiFluidMHD/AUSMFluxMultiFluid.hh"
#include "FiniteVolumeMultiFluidMHD/DriftWaves2DHalfTwoFluid.hh"
#include "FiniteVolumeMultiFluidMHD/HartmannSourceTerm.hh"

//////////////////////////////////////////////////////////////////////////////

using namespace COOLFluiD::Framework;
using namespace COOLFluiD::Common;
using namespace COOLFluiD::Config;
using namespace COOLFluiD::Physics::MHD;
using namespace COOLFluiD::Physics::Maxwell;
using namespace COOLFluiD::Physics::MultiFluidMHD;

//////////////////////////////////////////////////////////////////////////////

namespace COOLFluiD {

  namespace Numerics {

    namespace FiniteVolume {

//////////////////////////////////////////////////////////////////////////////


//Provider for AUSMPlusUpFlux with Source
#define FVMCC_MULTIFLUIDMHD_RHS_PROV_AUSMPLUSUP_SOURCE(__dim__,__half__,__svars__,__uvars__,__sourceterm__,__nbBThreads__,__providerName__) \
MethodCommandProvider<FVMCC_ComputeSourceRHSCell<AUSMPlusUpFluxMultiFluid<MultiFluidMHDVarSet<Maxwell##__dim__##ProjectionVarSet> >, \
			              VarSetListT<EulerMFMHD##__dim__##__half__##__svars__##T, EulerMFMHD##__dim__##__half__##__uvars__##T>, \
				      __sourceterm__<MultiFluidMHDVarSet<Maxwell##__dim__##ProjectionVarSet> >, \
				      LeastSquareP1PolyRec##__dim__ , BarthJesp, __nbBThreads__>, \
		      CellCenterFVMData, FiniteVolumeCUDAModule>	\
fvmcc_RhsMultiFluidMHDAUSMPlusUp##__dim__##__half__##__svars__##__uvars__##__sourceterm__##__nbBThreads__##Provider(__providerName__);

// 48 block threads (default)
FVMCC_MULTIFLUIDMHD_RHS_PROV_AUSMPLUSUP_SOURCE(2D,Half,Cons,RhoiViTi,DriftWaves2DHalfTwoFluid,48,"CellAUSMPlusUpEulerMFMHD2DHalfRhoiViTiDriftWavesTwoFluid")
FVMCC_MULTIFLUIDMHD_RHS_PROV_AUSMPLUSUP_SOURCE(2D,,Cons,RhoiViTi,HartmannSourceTerm,48,"CellAUSMPlusUpEulerMFMHD2DHalfRhoiViTiHartmann")
#undef FVMCC_MULTIFLUIDMHD_RHS_PROV_AUSMPLUSUP_SOURCE

//////////////////////////////////////////////////////////////////////////////

template <typename PHYS>
HOST_DEVICE inline void setState(CFreal* state, CFreal* statePtr, 
				 CFreal* node, CFreal* nodePtr)
{
  // copy the state node data to shared memory
  for (CFuint i = 0; i < PHYS::DIM; ++i) {node[i] = nodePtr[i];}
  // copy the state data to shared memory
  for (CFuint i = 0; i < PHYS::NBEQS; ++i) {state[i] = statePtr[i];} 
}
      
//////////////////////////////////////////////////////////////////////////////
      
template <typename PHYS>
HOST_DEVICE inline void setFaceNormal(FluxData<PHYS>* fd, CFreal* normal)
{
  CudaEnv::CFVecSlice<CFreal,PHYS::DIM> n(normal);
  const CFreal area = n.norm2();
  fd->setFaceArea(area);
  const CFreal ovArea = 1./area;
  CudaEnv::CFVecSlice<CFreal,PHYS::DIM> un(fd->getUnitNormal());
  for (CFuint i = 0; i < PHYS::DIM; ++i) {
    un[i] = n[i]*ovArea;
  }
}
      
//////////////////////////////////////////////////////////////////////////////

template <typename PHYS, typename PTR>
HOST_DEVICE void setFluxData(const CFuint f, const CFint stype, 
			     const CFuint stateID, const CFuint cellID, 
			     KernelData<CFreal>* kd, FluxData<PHYS>* fd,
			     PTR cellFaces)
{
  fd->setStateID(RIGHT, stateID);
  CFreal* statePtrR = (stype > 0) ? &kd->states[stateID*PHYS::NBEQS] : &kd->ghostStates[stateID*PHYS::NBEQS];  
  CFreal* nodePtrR = (stype > 0) ? &kd->centerNodes[stateID*PHYS::DIM] : &kd->ghostNodes[stateID*PHYS::DIM];  
  setState<PHYS>(fd->getState(RIGHT), statePtrR, fd->getNode(RIGHT), nodePtrR);
  
  fd->setIsBFace(stype < 0);
  fd->setStateID(LEFT, cellID);
  const CFuint faceID = cellFaces[f*kd->nbCells + cellID];
  fd->setIsOutward(kd->isOutward[faceID] == cellID);
  
  CFreal* statePtrL = &kd->states[cellID*PHYS::NBEQS];
  CFreal* nodePtrL = &kd->centerNodes[cellID*PHYS::DIM];
  setState<PHYS>(fd->getState(LEFT), statePtrL, fd->getNode(LEFT), nodePtrL);
  setFaceNormal<PHYS>(fd, &kd->normals[faceID*PHYS::DIM]);
}

//////////////////////////////////////////////////////////////////////////////

template <typename T, CFuint SIZE>
void print(const std::string& name, T* array) 
{
  CFLog(INFO, name << " = \t");
  for (CFuint i = 0; i < SIZE; ++i) {
    CFLog(INFO, array[i] << " ");
  }
  CFLog(INFO, "\n");
}

//////////////////////////////////////////////////////////////////////////////

template <typename MODEL>
HOST_DEVICE void computeFaceCentroid(const CellData::Itr* cell, const CFuint faceIdx, 
				     const CFreal* nodes, CFreal* midFaceCoord)
{  
  CudaEnv::CFVecSlice<CFreal, MODEL::DIM> coord(midFaceCoord);
  coord = 0.;
  const CFuint nbFaceNodes = cell->getNbFaceNodes(faceIdx);
  const CFreal ovNbFaceNodes = 1./(static_cast<CFreal>(nbFaceNodes));
  for (CFuint n = 0; n < nbFaceNodes; ++n) {
    const CFuint cellNodeID = cell->getNodeID(faceIdx, n);
    const CFuint nodeID = cell->getNodeID(faceIdx,n);
    const CFreal* faceNode = &nodes[nodeID*MODEL::DIM];
    for (CFuint d = 0; d < MODEL::DIM; ++d) {
      coord[d] += faceNode[d];
    }
  }
  coord *= ovNbFaceNodes;
}

//////////////////////////////////////////////////////////////////////////////

template <typename PHYS, typename POLYREC>
__global__ void computeGradientsKernel(typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
				       const CFuint nbCells,
				       CFreal* states, 
				       CFreal* nodes,
				       CFreal* centerNodes,
				       CFreal* ghostStates,
				       CFreal* ghostNodes,
				       CFreal* uX,
				       CFreal* uY,
				       CFreal* uZ,
				       CFreal* limiter,
				       CFreal* updateCoeff, 
				       CFreal* rhs,
				       CFreal* normals,
				       CFint* isOutward,
				       const CFuint* cellInfo,
				       const CFuint* cellStencil,
				       const CFuint* cellFaces,
				       const CFuint* cellNodes,
				       const CFint*  neighborTypes,
				       const Framework::CellConn* cellConn)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
  // dcor->init(gdcor);
  
  if (cellID < nbCells) {    
    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
			   rhs, normals, uX, uY, uZ, isOutward);
    
    // compute and store cell gradients at once 
    POLYREC polyRec(dcor);
    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
    CellData::Itr cell = cells.getItr(cellID);
    polyRec.computeGradients(&states[cellID*PHYS::NBEQS], &centerNodes[cellID*PHYS::DIM], &kd, &cell);
  }
}
      
//////////////////////////////////////////////////////////////////////////////

template <typename PHYS, typename POLYREC, typename LIMITER>
__global__ void computeLimiterKernel(typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
				     typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
				     const CFuint nbCells,
				     CFreal* states, 
				     CFreal* nodes,
				     CFreal* centerNodes,
				     CFreal* ghostStates,
				     CFreal* ghostNodes,
				     CFreal* uX,
				     CFreal* uY,
				     CFreal* uZ,
				     CFreal* limiter,
				     CFreal* updateCoeff, 
				     CFreal* rhs,
				     CFreal* normals,
				     CFint* isOutward,
				     const CFuint* cellInfo,
				     const CFuint* cellStencil,
				     const CFuint* cellFaces,
				     const CFuint* cellNodes,
				     const CFint*  neighborTypes,
				     const Framework::CellConn* cellConn)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  // __shared__ typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE> s_dcol[32];
  // typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol = &s_dcol[threadIdx.x];
  // dcol->init(gdcol);
  
  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
  // dcor->init(gdcor);
  
  if (cellID < nbCells) {    
    // compute all cell quadrature points at once (size of this array is overestimated)
    CFreal midFaceCoord[PHYS::DIM*PHYS::DIM*2];
    
    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
    CellData::Itr cell = cells.getItr(cellID);
    const CFuint nbFacesInCell = cell.getNbFacesInCell();
    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
      computeFaceCentroid<PHYS>(&cell, f, nodes, &midFaceCoord[f*PHYS::DIM]);
    }
    
    // compute cell-based limiter at once
    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
			   rhs, normals, uX, uY, uZ, isOutward);
    LIMITER limt(dcol);
    
    if (dcor->currRes > dcor->limitRes && (dcor->limitIter > 0 && dcor->currIter < dcor->limitIter)) {	
      limt.limit(&kd, &cell, &midFaceCoord[0], &limiter[cellID*PHYS::NBEQS]);
    }
    else {
      if (!dcor->freezeLimiter) {
	// historical modification of the limiter
	CudaEnv::CFVec<CFreal,PHYS::NBEQS> tmpLimiter;
	limt.limit(&kd, &cell, &midFaceCoord[0], &tmpLimiter[0]);
	CFuint currID = cellID*PHYS::NBEQS;
	for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar, ++currID) {
	  limiter[currID] = min(tmpLimiter[iVar],limiter[currID]);
	}
      }
    }
  }
}
  
//////////////////////////////////////////////////////////////////////////////
             
template <typename SCHEME, typename POLYREC>
__global__ void computeFluxKernel(typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof,
				  typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
				  typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
				  const CFuint nbCells,
				  CFreal* states, 
				  CFreal* nodes,
				  CFreal* centerNodes,
				  CFreal* ghostStates,
				  CFreal* ghostNodes,
				  CFreal* uX,
				  CFreal* uY,
				  CFreal* uZ,
				  CFreal* limiter,
				  CFreal* updateCoeff, 
				  CFreal* rhs,
				  CFreal* normals,
				  CFint* isOutward,
				  const CFuint* cellInfo,
				  const CFuint* cellStencil,
				  const CFuint* cellFaces,
				  const CFuint* cellNodes,
				  const CFint*  neighborTypes,
				  const Framework::CellConn* cellConn)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  // __shared__ typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE> s_dcof[32];
  // typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof = &s_dcof[threadIdx.x];
  // dcof->init(gdcof);
  
  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
  // dcor->init(gdcor);
  
  // __shared__ typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE> s_dcop[32];
  // typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop = &s_dcop[threadIdx.x];
  // dcop->init(gdcop);
  
  if (cellID < nbCells) {
    // reset the rhs and update coefficients to 0
    CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> res(&rhs[cellID*SCHEME::MODEL::NBEQS]);
    res = 0.;
    updateCoeff[cellID] = 0.;
    
    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
			   rhs, normals, uX, uY, uZ, isOutward);
    
    // compute and store cell gradients at once 
    POLYREC polyRec(dcor);
    SCHEME fluxScheme(dcof);
    CFreal midFaceCoord[SCHEME::MODEL::DIM*SCHEME::MODEL::DIM*2];
    FluxData<typename SCHEME::MODEL> currFd; currFd.initialize();
    typename SCHEME::MODEL pmodel(dcop);
    
    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
    CellData::Itr cell = cells.getItr(cellID);
    
    // compute the fluxes
    const CFuint nbFacesInCell = cell.getNbActiveFacesInCell();
    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
      const CFint stype = cell.getNeighborType(f);
      
      if (stype != 0) { // skip all partition faces
	// set all flux data for the current face
	const CFuint stateID = cell.getNeighborID(f);
	setFluxData(f, stype, stateID, cellID, &kd, &currFd, cellFaces);
	
	// compute face quadrature points (centroid)
	CFreal* faceCenters = &midFaceCoord[f*SCHEME::MODEL::DIM];
	computeFaceCentroid<typename SCHEME::MODEL>(&cell, f, nodes, faceCenters);
	
	// extrapolate solution on quadrature points on both sides of the face
	polyRec.extrapolateOnFace(&currFd, faceCenters, uX, uY, uZ, limiter);
	
	// compute the convective flux across the face
	fluxScheme(&currFd, &pmodel);
	
	// update the residual
	CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> ress(currFd.getResidual());
	res -= ress;
	
	// update the update coefficient
	updateCoeff[cellID] += currFd.getUpdateCoeff();
      }
    }
  }
}
 


//////////////////////////////////////////////////////////////////////////////

template <typename SOURCE>
__global__ void computeSource(typename SOURCE::BASE::template DeviceConfigOptions<NOTYPE>* dcos,
				  typename SOURCE::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
				  const CFuint nbCells,
				  CFreal* states, 
                                  CFreal* volumes,
				  CFreal* nodes,
				  CFreal* centerNodes,
				  CFreal* ghostStates,
				  CFreal* ghostNodes,
				  CFreal* uX,
				  CFreal* uY,
				  CFreal* uZ,
				  CFreal* limiter,
				  CFreal* updateCoeff, 
				  CFreal* rhs,
				  CFreal* normals,
				  CFint* isOutward,
				  const CFuint* cellInfo,
				  const CFuint* cellStencil,
				  const CFuint* cellFaces,
				  const CFuint* cellNodes,
				  const CFint*  neighborTypes,
				  const Framework::CellConn* cellConn,
				  CFreal ResFactor, bool IsAxisymmetric)
{
  // each thread takes care of computing the source for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;

  const CFuint nbEqs = SOURCE::MODEL::NBEQS;
  CudaEnv::CFVec<CFreal,SOURCE::MODEL::NBEQS> source;
  source = 0.;

  SOURCE Source(dcos);
  typename SOURCE::MODEL pmodel(dcop);

  CudaEnv::CFVecSlice<CFreal,SOURCE::MODEL::NBEQS> state(&states[cellID*SOURCE::MODEL::NBEQS]);
  Source(&state[0], &pmodel, &source[0]);
      
  CFreal invR = 1.0;
  if (IsAxisymmetric) {     
    //invR /= abs(currCell->getState(0)->getCoordinates()[YY]);  //Not implemented
  }


  for (CFuint iEq = 0; iEq < nbEqs; ++iEq) { 
     rhs[iEq] += ResFactor*source[iEq]*invR;   
  }
}



 
//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename SOURCE, typename POLYREC, typename LIMITER>
void computeFluxSourceCPU(typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof,
		    typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
		    typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
		    typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
                    typename SOURCE::BASE::template DeviceConfigOptions<NOTYPE>* dcos,
		    const CFuint nbCells,
		    CFreal* states, 
                    CFreal* volumes,
		    CFreal* nodes,
		    CFreal* centerNodes,
		    CFreal* ghostStates,
		    CFreal* ghostNodes,
		    CFreal* uX,
		    CFreal* uY,
		    CFreal* uZ,
		    CFreal* limiter,
		    CFreal* updateCoeff, 
		    CFreal* rhs,
		    CFreal* normals,
		    CFint* isOutward,
		    const CFuint* cellInfo,
		    const CFuint* cellStencil,
		    const CFuint* cellFaces,
		    const CFuint* cellNodes,
		    const CFint* neighborTypes,
		    const Framework::CellConn* cellConn,
                    CFreal ResFactor, bool IsAxisymmetric)
{ 
  typedef typename SCHEME::MODEL PHYS;
  
  FluxData<PHYS> fd; fd.initialize();
  FluxData<PHYS>* currFd = &fd;
  cf_assert(currFd != CFNULL);
  SCHEME fluxScheme(dcof);
  POLYREC polyRec(dcor);
  LIMITER limt(dcol);
  PHYS pmodel(dcop);
  
  CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
  KernelData<CFreal> kd(nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
			rhs, normals, uX, uY, uZ, isOutward);
  
  CFreal midFaceCoord[PHYS::DIM*PHYS::DIM*2];
  CudaEnv::CFVec<CFreal,PHYS::NBEQS> tmpLimiter;
  
  CudaEnv::CFVec<CFreal,PHYS::NBEQS> source;
  SOURCE Source(dcos);

  // compute the cell-based gradients
  for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
    CellData::Itr cell = cells.getItr(cellID);
    polyRec.computeGradients(&states[cellID*PHYS::NBEQS], &centerNodes[cellID*PHYS::DIM], &kd, &cell);
  }
  
  // compute the cell based limiter
  // for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
  for (CellData::Itr cell = cells.begin(); cell <= cells.end(); ++cell) {
    // compute all cell quadrature points at once (size of this array is overestimated)
    const CFuint nbFacesInCell = cell.getNbFacesInCell();
    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
      computeFaceCentroid<PHYS>(&cell, f, nodes, &midFaceCoord[f*PHYS::DIM]);
    }
    
    const CFuint cellID = cell.getCellID();
    if (dcor->currRes > dcor->limitRes && (dcor->limitIter > 0 && dcor->currIter < dcor->limitIter)) {	
      // compute cell-based limiter
      limt.limit(&kd, &cell, &midFaceCoord[0], &limiter[cellID*PHYS::NBEQS]);
    }
    else {
      if (!dcor->freezeLimiter) {
	// historical modification of the limiter
	limt.limit(&kd, &cell, &midFaceCoord[0], &tmpLimiter[0]);
	CFuint currID = cellID*PHYS::NBEQS;
	for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar, ++currID) {
	  limiter[currID] = min(tmpLimiter[iVar],limiter[currID]);
	}
      }
    }
  }
  
  // compute the fluxes
  for (CellData::Itr cell = cells.begin(); cell <= cells.end(); ++cell) {
    // reset the rhs and update coefficients to 0
    const CFuint cellID = cell.getCellID();
    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> res(&rhs[cellID*PHYS::NBEQS]);
    res = 0.;
    updateCoeff[cellID] = 0.;
    
    const CFuint nbFacesInCell = cell.getNbActiveFacesInCell();
    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
      const CFint stype = cell.getNeighborType(f);
      
      if (stype != 0) { // skip all partition faces
	const CFuint stateID =  cell.getNeighborID(f);
	setFluxData(f, stype, stateID, cellID, &kd, currFd, cellFaces);
	
	// compute face quadrature points (centroid)
	CFreal* faceCenters = &midFaceCoord[f*PHYS::DIM];
	computeFaceCentroid<PHYS>(&cell, f, nodes, faceCenters);
	
	// extrapolate solution on quadrature points on both sides of the face
	polyRec.extrapolateOnFace(currFd, faceCenters, uX, uY, uZ, limiter);
	fluxScheme(currFd, &pmodel); // compute the convective flux across the face
	
	for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) {
	  const CFreal value = currFd->getResidual()[iEq];
	  res[iEq] -= value;  // update the residual 
	}
	
	// update the update coefficient
	updateCoeff[cellID] += currFd->getUpdateCoeff();
      }
    }

 
    //Source computation
    source = 0.;
    
 
    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> state(&states[cellID*PHYS::NBEQS]);
    Source(&state[0], &pmodel, &source[0]);

    CFreal invR = 1.0;
    if (IsAxisymmetric) {     
      //invR /= abs(currCell->getState(0)->getCoordinates()[YY]);  
    }
    CFreal factor = invR*volumes[cellID]*ResFactor;     

    source *= factor;
    for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) { 
      res[iEq] += source[iEq];   
    }
    
    /////////////////////////////////////////////


  }
}

//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename PHYSICS, typename SOURCE,typename POLYREC, typename LIMITER, CFuint NB_BLOCK_THREADS>
void FVMCC_ComputeSourceRHSCell<SCHEME,PHYSICS,SOURCE,POLYREC,LIMITER,NB_BLOCK_THREADS>::execute()
{
  using namespace COOLFluiD::Framework;
  using namespace COOLFluiD::Common;
  
  CFTRACEBEGIN;
  
  CFLog(VERBOSE, "FVMCC_ComputeSourceRHSCell::execute() START\n");
  
  initializeComputationRHS();



  const CFuint nbCells = socket_states.getDataHandle().size();
  cf_assert(nbCells > 0);
  DataHandle<CFreal> updateCoeff = socket_updateCoeff.getDataHandle();
  DataHandle<CFreal> rhs = socket_rhs.getDataHandle();
  DataHandle<CFreal> normals = socket_normals.getDataHandle();
  DataHandle<CFint> isOutward = socket_isOutward.getDataHandle();  
  
  SafePtr<SCHEME> lf  = getMethodData().getFluxSplitter().d_castTo<SCHEME>();
  SafePtr<POLYREC> pr = getMethodData().getPolyReconstructor().d_castTo<POLYREC>();
  SafePtr<LIMITER> lm = getMethodData().getLimiter().d_castTo<LIMITER>();
  SafePtr<typename PHYSICS::PTERM> phys = PhysicalModelStack::getActive()->getImplementor()->
    getConvectiveTerm().d_castTo<typename PHYSICS::PTERM>();
  
  typedef typename SCHEME::template DeviceFunc<GPU, PHYSICS> FluxScheme;  
  typedef typename POLYREC::template DeviceFunc<PHYSICS> PolyRec;  
  typedef typename LIMITER::template DeviceFunc<PHYSICS> Limiter;  
  
  //Added for Source
  SelfRegistPtr<SOURCE> ls1  = (*this->getMethodData().getSourceTermComputer())[0].template d_castTo<SOURCE>();  //Only valid if there is only one source term!!
  SafePtr<SOURCE> ls = ls1.getPtr();
  typedef typename SOURCE::template DeviceFunc<GPU, PHYSICS> SourceTerm; 


  if (m_onGPU) {

    CudaEnv::CudaTimer& timer = CudaEnv::CudaTimer::getInstance();
    timer.start();
    
    // copy of data that change at every iteration
    socket_states.getDataHandle().getGlobalArray()->put();
    socket_volumes.getDataHandle().getLocalArray()->put(); 
    m_ghostStates.put();
     
    CFLog(VERBOSE, "FVMCC_ComputeSourceRHSCell::execute() => CPU-->GPU data transfer took " << timer.elapsed() << " s\n");
    timer.start();
    
    ConfigOptionPtr<POLYREC, NOTYPE, GPU> dcor(pr);
    ConfigOptionPtr<LIMITER, NOTYPE, GPU> dcol(lm);
    ConfigOptionPtr<SCHEME,  NOTYPE, GPU> dcof(lf);
    ConfigOptionPtr<typename PHYSICS::PTERM, NOTYPE, GPU> dcop(phys);
    
    //Added for Source    
    ConfigOptionPtr<SOURCE, NOTYPE, GPU> dcos(ls);



    const CFuint blocksPerGrid = CudaEnv::CudaDeviceManager::getInstance().getBlocksPerGrid(nbCells);
    const CFuint nThreads = CudaEnv::CudaDeviceManager::getInstance().getNThreads();
    
    //dim3 blocks(m_nbBlocksPerGridX, m_nbBlocksPerGridY);
    
    //hipFuncSetCacheConfig(reinterpret_cast<const void*>("computeGradientsKernel"), hipFuncCachePreferL1);
    
        
    // compute the cell-based gradients
    computeGradientsKernel<PHYSICS, PolyRec> <<<blocksPerGrid,nThreads>>> 
      (dcor.getPtr(),
       nbCells,
       socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
       socket_nodes.getDataHandle().getGlobalArray()->ptrDev(),
       m_centerNodes.ptrDev(), 
       m_ghostStates.ptrDev(),
       m_ghostNodes.ptrDev(),
       socket_uX.getDataHandle().getLocalArray()->ptrDev(),
       socket_uY.getDataHandle().getLocalArray()->ptrDev(),
       socket_uZ.getDataHandle().getLocalArray()->ptrDev(),
       socket_limiter.getDataHandle().getLocalArray()->ptrDev(),
       updateCoeff.getLocalArray()->ptrDev(), 
       rhs.getLocalArray()->ptrDev(),
       normals.getLocalArray()->ptrDev(),
       isOutward.getLocalArray()->ptrDev(),
       m_cellInfo.ptrDev(),
       m_cellStencil.ptrDev(),
       m_cellFaces->getPtr()->ptrDev(),
       m_cellNodes->getPtr()->ptrDev(),
       m_neighborTypes.ptrDev(),
       m_cellConn.ptrDev());
    
    CFLog(VERBOSE, "FVMCC_ComputeSourceRHSCell::execute() => computeGradientsKernel took " << timer.elapsed() << " s\n");
    
    timer.start();
    
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>("computeLimiterKernel"), hipFuncCachePreferL1);
    
    // compute the limiter in each cell
    computeLimiterKernel<PHYSICS, PolyRec, Limiter> <<<blocksPerGrid,nThreads>>> 
      (dcol.getPtr(),
       dcor.getPtr(),
       nbCells,
       socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
       socket_nodes.getDataHandle().getGlobalArray()->ptrDev(),
       m_centerNodes.ptrDev(), 
       m_ghostStates.ptrDev(),
       m_ghostNodes.ptrDev(),
       socket_uX.getDataHandle().getLocalArray()->ptrDev(),
       socket_uY.getDataHandle().getLocalArray()->ptrDev(),
       socket_uZ.getDataHandle().getLocalArray()->ptrDev(),
       socket_limiter.getDataHandle().getLocalArray()->ptrDev(),
       updateCoeff.getLocalArray()->ptrDev(), 
       rhs.getLocalArray()->ptrDev(),
       normals.getLocalArray()->ptrDev(),
       isOutward.getLocalArray()->ptrDev(),
       m_cellInfo.ptrDev(),
       m_cellStencil.ptrDev(),
       m_cellFaces->getPtr()->ptrDev(),
       m_cellNodes->getPtr()->ptrDev(),
       m_neighborTypes.ptrDev(),
       m_cellConn.ptrDev());
    
    CFLog(VERBOSE, "FVMCC_ComputeSourceRHSCell::execute() => computeLimiterKernel took " << timer.elapsed() << " s\n");
    
    timer.start();
    
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>("computeFluxKernel"), hipFuncCachePreferL1);
    
    // compute the convective flux in each cell
    computeFluxKernel<FluxScheme, PolyRec> <<<blocksPerGrid,nThreads>>> 
      (dcof.getPtr(),
       dcor.getPtr(),
       dcop.getPtr(),
       nbCells,
       socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
       socket_nodes.getDataHandle().getGlobalArray()->ptrDev(),
       m_centerNodes.ptrDev(), 
       m_ghostStates.ptrDev(),
       m_ghostNodes.ptrDev(),
       socket_uX.getDataHandle().getLocalArray()->ptrDev(),
       socket_uY.getDataHandle().getLocalArray()->ptrDev(),
       socket_uZ.getDataHandle().getLocalArray()->ptrDev(),
       socket_limiter.getDataHandle().getLocalArray()->ptrDev(),
       updateCoeff.getLocalArray()->ptrDev(), 
       rhs.getLocalArray()->ptrDev(),
       normals.getLocalArray()->ptrDev(),
       isOutward.getLocalArray()->ptrDev(),
       m_cellInfo.ptrDev(),
       m_cellStencil.ptrDev(),
       m_cellFaces->getPtr()->ptrDev(),
       m_cellNodes->getPtr()->ptrDev(),
       m_neighborTypes.ptrDev(),
       m_cellConn.ptrDev());
    
    CFLog(VERBOSE, "FVMCC_ComputeSourceRHSCell::execute() => computeFluxKernel took " << timer.elapsed() << " s\n");

    timer.start();
    CFLog(VERBOSE, "FVMCC_ComputeRHS::execute() => before computeSourceTerm()\n");

    bool IsAxisymmetric = this->getMethodData().isAxisymmetric(); //Default = false
    CFreal ResFactor = this->getMethodData().getResFactor(); //Default = 1

    computeSource<SourceTerm> <<<blocksPerGrid,nThreads>>> 
      (dcos.getPtr(),
       dcop.getPtr(),
       nbCells,
       socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
       socket_volumes.getDataHandle().getLocalArray()->ptrDev(),
       socket_nodes.getDataHandle().getGlobalArray()->ptrDev(),
       m_centerNodes.ptrDev(), 
       m_ghostStates.ptrDev(),
       m_ghostNodes.ptrDev(),
       socket_uX.getDataHandle().getLocalArray()->ptrDev(),
       socket_uY.getDataHandle().getLocalArray()->ptrDev(),
       socket_uZ.getDataHandle().getLocalArray()->ptrDev(),
       socket_limiter.getDataHandle().getLocalArray()->ptrDev(),
       updateCoeff.getLocalArray()->ptrDev(), 
       rhs.getLocalArray()->ptrDev(),
       normals.getLocalArray()->ptrDev(),
       isOutward.getLocalArray()->ptrDev(),
       m_cellInfo.ptrDev(),
       m_cellStencil.ptrDev(),
       m_cellFaces->getPtr()->ptrDev(),
       m_cellNodes->getPtr()->ptrDev(),
       m_neighborTypes.ptrDev(),
       m_cellConn.ptrDev(),
       ResFactor, IsAxisymmetric);

    CFLog(VERBOSE, "FVMCC_ComputeRHS::execute() => computeSourceTerm took " << timer.elapsed() << "\n");

    timer.start();
    rhs.getLocalArray()->get();
    updateCoeff.getLocalArray()->get();
    CFLog(VERBOSE, "FVMCC_ComputeSourceRHSCell::execute() => GPU-->CPU data transfer took " << timer.elapsed() << " s\n");
  }
  else {
    // AL: useful fo debugging
    // for (CFuint i = 0; i <  m_ghostStates.size()/9; ++i) {
    //   std::cout.precision(12); std::cout << "g" << i << " => ";
    //   for (CFuint j = 0; j < 9; ++j) {
    // 	std::cout << m_ghostStates[i*9+j] << " ";
    //   }
    //   std::cout << "\n";
    // }
    // for (CFuint i = 0; i <  socket_states.getDataHandle().size(); ++i) {
    //   std::cout.precision(12); std::cout << i << " => "<< *socket_states.getDataHandle()[i] <<"\n";
    // }
    
    ConfigOptionPtr<SCHEME>  dcof(lf);
    ConfigOptionPtr<POLYREC> dcor(pr);
    ConfigOptionPtr<LIMITER> dcol(lm);
    ConfigOptionPtr<typename PHYSICS::PTERM> dcop(phys);
    ConfigOptionPtr<SOURCE> dcos(ls);

    bool IsAxisymmetric = this->getMethodData().isAxisymmetric(); //Default = false
    CFreal ResFactor = this->getMethodData().getResFactor(); //Default = 1

    computeFluxSourceCPU<FluxScheme, SourceTerm, PolyRec, Limiter>
      (dcof.getPtr(),
       dcor.getPtr(),
       dcol.getPtr(),
       dcop.getPtr(),
       dcos.getPtr(),
       nbCells,
       socket_states.getDataHandle().getGlobalArray()->ptr(), 
       socket_volumes.getDataHandle().getLocalArray()->ptr(),
       socket_nodes.getDataHandle().getGlobalArray()->ptr(),
       m_centerNodes.ptr(), 
       m_ghostStates.ptr(),
       m_ghostNodes.ptr(),
       socket_uX.getDataHandle().getLocalArray()->ptr(),
       socket_uY.getDataHandle().getLocalArray()->ptr(),
       socket_uZ.getDataHandle().getLocalArray()->ptr(),
       socket_limiter.getDataHandle().getLocalArray()->ptr(),
       updateCoeff.getLocalArray()->ptr(), 
       rhs.getLocalArray()->ptr(),
       normals.getLocalArray()->ptr(),
       isOutward.getLocalArray()->ptr(),
       m_cellInfo.ptr(),
       m_cellStencil.ptr(),
       m_cellFaces->getPtr()->ptr(),
       m_cellNodes->getPtr()->ptr(),
       m_neighborTypes.ptr(),
       m_cellConn.ptr(),
       ResFactor, IsAxisymmetric);
  }
  
// for (int i = 0; i < updateCoeff.size(); ++i) {
//      std::cout << "updateCoeff[" << i << "] = " << updateCoeff[i]  << std::endl;
//       /* std::cout << "rhs[" << i << "] = ";
//        for (int j = 0; j < 9; ++j) {
//          std::cout << rhs[i*9+j] << " ";
//        }
//        std::cout << std::endl;*/
// } 
//   abort();
  
  finalizeComputationRHS();
  
  CFLog(VERBOSE, "FVMCC_ComputeSourceRHSCell::execute() END\n");
  
  CFTRACEEND;
}

//////////////////////////////////////////////////////////////////////////////

    } // namespace FiniteVolume
    
  } // namespace Numerics

} // namespace COOLFluiD
