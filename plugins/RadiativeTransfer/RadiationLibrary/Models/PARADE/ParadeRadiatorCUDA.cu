#include "hip/hip_runtime.h"
#include <fstream>

#include "RadiativeTransfer/RadiationLibrary/Models/PARADE/ParadeRadiatorCUDA.hh"
#include "RadiativeTransfer/RadiationLibrary/RadiationPhysicsHandler.hh"

#include "Common/CFLog.hh"
#include "Common/DebugFunctions.hh"
#include "Common/CFPrintContainer.hh"
#include "Framework/CudaTimer.hh"

#include "Environment/ObjectProvider.hh"
#include "Environment/CFEnv.hh"
#include "Environment/FileHandlerOutput.hh"
#include "Environment/FileHandlerInput.hh"
#include "Environment/SingleBehaviorFactory.hh"
#include "Environment/DirPaths.hh"
#include "Common/BadValueException.hh"
#include "Common/Stopwatch.hh"
#include "Common/PEFunctions.hh"
#include "Framework/MeshData.hh"
#include "Framework/PhysicalChemicalLibrary.hh"
#include "Framework/PhysicalModel.hh"
#include "Framework/PhysicalConsts.hh"

//////////////////////////////////////////////////////////////////////////////

using namespace std;
using namespace COOLFluiD::Framework;
using namespace COOLFluiD::MathTools;
using namespace COOLFluiD::Common;
using namespace COOLFluiD::Environment;

//////////////////////////////////////////////////////////////////////////////

namespace COOLFluiD {

  namespace RadiativeTransfer {

//////////////////////////////////////////////////////////////////////////////

Environment::ObjectProvider<ParadeRadiatorCUDA,
			    Radiator,
			    RadiativeTransferModule,
			    1>
paradeRadiatorCUDAProvider("ParadeRadiatorCUDA");

//////////////////////////////////////////////////////////////////////////////

__global__ void computeCellBinsKernel(const CFuint isEquil,
				      const CFuint nbPoints, 
				      const CFuint nbCells, 
				      const CFuint nbBinsre,
				      const CFuint testID,
				      const CFreal dWavIn,
				      const CFreal* vctBins,
				      CFreal* data,
				      CFreal* alpha_bin, 
				      CFreal* emission_bin,
				      CFreal* B_binCurr)
{
  const CFuint idx = threadIdx.x + blockIdx.x*blockDim.x;
  const CFuint cellID = idx%nbCells;
  const CFuint pointID  = idx/nbPoints;
  const CFuint sizeLoop = nbCells*nbPoints;
  
  // fill the alpha and emission arrayas for each local cell in parallel simulations
  // each row gives all the bins for a cell
  if (idx < sizeLoop) {
    ParadeRadiator::DeviceFunc df;
    const CFreal temp = 300.; // AL: here the equilibrium case is basically not supported
    df.computeCellBins<GPU>(isEquil, nbPoints, pointID, cellID, nbBinsre, testID, temp, dWavIn,
			    vctBins, data, alpha_bin, emission_bin, B_binCurr);
  }
}
    
//////////////////////////////////////////////////////////////////////////////

void ParadeRadiatorCUDA::defineConfigOptions(Config::OptionList& options)
{
}
  
//////////////////////////////////////////////////////////////////////////////

ParadeRadiatorCUDA::ParadeRadiatorCUDA(const std::string& name) :
  ParadeRadiator(name)
{
  addConfigOptionsTo(this);
}
  
//////////////////////////////////////////////////////////////////////////////

ParadeRadiatorCUDA::~ParadeRadiatorCUDA()
{
}

//////////////////////////////////////////////////////////////////////////////
      
void ParadeRadiatorCUDA::setup()
{
  CFLog(VERBOSE, "ParadeRadiatorCUDA::setup() => START\n");
  
  ParadeRadiator::setup();
    
  CFLog(VERBOSE, "ParadeRadiatorCUDA::setup() => END\n");
}
  
//////////////////////////////////////////////////////////////////////////////
            
void ParadeRadiatorCUDA::unsetup()
{
  CFLog(VERBOSE, "ParadeRadiatorCUDA::unsetup() => START\n");
  
  ParadeRadiator::unsetup();
  
  CFLog(VERBOSE, "ParadeRadiatorCUDA::unsetup() => END\n");
}
  
/////////////////////////////////////////////////////////////////////////////

void ParadeRadiatorCUDA::computeBinning()
{    
  CFLog(VERBOSE, "ParadeRadiatorCUDA::computeBinning() => START\n");

  CudaEnv::CudaTimer& timer = CudaEnv::CudaTimer::getInstance();
  
  CFuint totalNbCells = this->m_pstates->getSize();
  const CFuint nbCols = m_nbPoints*3;
  CFuint nbCells = m_data.size()/nbCols;
  cf_assert(m_saveMemory && nbCells <= totalNbCells);
  
  MPIError::getInstance().check
    ("MPI_Allreduce", "ParadeRadiatorCUDA::computeBinning()",
     MPI_Allreduce(&nbCells, &totalNbCells, 1, MPIStructDef::getMPIType(&nbCells), 
		   MPI_SUM, PE::GetPE().GetCommunicator(m_namespace)));
  cf_assert(totalNbCells == m_pstates->getSize());
  
  //Function to calculate the binning algorithm with absorption & emission coefficients
  const CFuint nbBins = m_nbBins;
  CFreal num_alphatot = 0.0;
  CFreal den_alphatot = 0.0;
  m_alphaav.resize(m_nbPoints, 0.);
  LocalArray<CFreal>::TYPE* num_alphatotVec = new LocalArray<CFreal>::TYPE(m_nbPoints, 0.);
  LocalArray<CFreal>::TYPE* den_alphatotVec = new LocalArray<CFreal>::TYPE(m_nbPoints, 0.);
  
  // offset for the ID from which the cells need to be counted 
  CFuint offsetStateID = 0;
  const CFuint nbCellsPerProc = totalNbCells/m_nbProc;
  for (CFuint rank = 0; rank < m_rank; ++rank) {
    offsetStateID += nbCellsPerProc;
  }
  
  timer.start();
  
  for(CFuint i=0;i<m_nbPoints;++i) {
    for(CFuint s=0;s<nbCells;++s) {
      const CFuint start = s*nbCols + i*3;
      const CFreal alpha = m_data[start+2];
      const CFreal epsilon = m_data[start+1];
      const CFreal Bs = epsilon/alpha;
      
      // Planck function if we are not in equilibrium
      //
      // CFreal h = 6.626070040e-34;     //SI units Js
      // CFuint c = 3e08; //SI units m/s
      // CFreal k_b = 1.3806485279e-23;  //SI units J/K
      //
      // T needs to be defined from the values extracted
      // const  B = (2*h*pow(c,2)/pow(m_data(j,i*3),5))*(1/(exp(h*c/(m_data(j,i*3)*k_b*T))-1));
      const CFuint stateID = s+offsetStateID;
      const CFreal volume = getCellVolume(m_pstates->getStateLocalID(stateID));
      const CFreal BsVolume = Bs*volume;
      const CFreal num_alpha_vol = alpha*BsVolume;
      const CFreal den_alpha_vol = BsVolume;
      num_alphatot += num_alpha_vol;
      den_alphatot += den_alpha_vol;
    }
    
    (*num_alphatotVec)[i] = num_alphatot;
    (*den_alphatotVec)[i] = den_alphatot;
  } 
  
  CFLog(INFO, "ParadeRadiatorCUDA::computeBinning() => num/den alpha took  " << timer.elapsed() << "s \n"); 
  
  /*computeNumDenAlphaKernel<<<blocks, threads>>>
    (nbPoints, nbCells, offsetStateID, 
    m_data.ptrDev(),
    num_alphatotVec.ptrDev(), 
    den_alphatotVec.ptrDev(), 
    m_radPhysicsHandlerPtr->getDataSockets()->volumes.getLocalArray()->ptrDev());*/
  
  // compute the total numerator and denominator per spectral point across all ranks
  vector<CFreal> num_alphatotGlobal(m_nbPoints, 0.);
  vector<CFreal> den_alphatotGlobal(m_nbPoints, 0.);
  MPIError::getInstance().check
    ("MPI_Allreduce", "ParadeRadiatorCUDA::computeBinning()",
     MPI_Allreduce(&(*num_alphatotVec)[0], &num_alphatotGlobal[0], m_nbPoints, 
		   MPIStructDef::getMPIType(&(*num_alphatotVec)[0]), 
		   MPI_SUM, PE::GetPE().GetCommunicator(m_namespace)));
  
  MPIError::getInstance().check
    ("MPI_Allreduce", "ParadeRadiatorCUDA::computeBinning()",
     MPI_Allreduce(&(*den_alphatotVec)[0], &den_alphatotGlobal[0], m_nbPoints, 
		   MPIStructDef::getMPIType(&(*den_alphatotVec)[0]), 
		   MPI_SUM, PE::GetPE().GetCommunicator(m_namespace)));
  
  cf_assert(m_alphaav.size() == m_nbPoints);
  for (CFuint i = 0; i < m_alphaav.size(); ++i) {
    cf_assert(std::abs(den_alphatotGlobal[i]) > 0.);
    m_alphaav[i] = num_alphatotGlobal[i] / den_alphatotGlobal[i];
    // CFLog(INFO, "m_alphaav[" << i << "] = " <<  m_alphaav[i] << "\n");
  }
  
  // free memory
  deletePtr(num_alphatotVec);
  deletePtr(den_alphatotVec);
  
  CFreal alphamin = m_alphaav[0];
  CFreal alphamax = m_alphaav[0];
  for(CFuint r=0;r<m_nbPoints;++r) {
    if(m_alphaav[r]<alphamin) {
      alphamin = m_alphaav[r];
    }
    if(m_alphaav[r]>alphamax) {
      alphamax = m_alphaav[r];
    }
  }
  
  CFLog(INFO,"ParadeLibrary::computeBinning () => [alphamin, alphamax] = [" 
	<< alphamin <<", " << alphamax << "]\n");
  
  // Logarithmic spacing
  //
  const CFreal alpha_minlog = std::log(alphamin);
  const CFreal alpha_maxlog = std::log(alphamax);
  
  CFLog(INFO,"ParadeLibrary::computeBinning () => [alpha_minlog, alpha_maxlog] = [" 
	<< alpha_minlog << ", " << alpha_maxlog << "]\n");
  
  const CFreal dy = (alpha_maxlog-alpha_minlog) / (nbBins-1);
  CFLog(VERBOSE,"ParadeLibrary::computeBinning () => dy = " << dy <<"\n");
  
  LocalArray<CFreal>::TYPE vctBins(0., nbBins);
  for(CFuint i = 0; i<nbBins; ++i) {
    vctBins[i] = std::exp(alpha_minlog + (dy * i));
    CFLog(VERBOSE,"ParadeLibrary::computeBinning () => vctBins(" << i << ") = " << vctBins[i] <<"\n");
  }
  
  // To search for minimum and maximum alpha
  CFreal alphamin_tot = m_alphaav[0];
  CFreal alphamax_tot = m_alphaav[0];
  for(CFuint i=0;i<m_nbPoints;++i) {
    for(CFuint s=0;s<nbCells;s++) {
      const CFreal alphaA = m_data[s*nbCols + i*3+2];
      if(alphaA < alphamin_tot) {alphamin_tot = alphaA;}
      if(alphaA > alphamax_tot) {alphamax_tot = alphaA;}
    }
  }
  
  if (m_saveMemory) {
    CFreal alphaMinTotLocal = alphamin_tot;
    CFreal alphaMaxTotLocal = alphamax_tot;
    CFreal alphaMinTotGlobal = 0.;
    CFreal alphaMaxTotGlobal = 0.;
    MPIError::getInstance().check
      ("MPI_Allreduce", "ParadeRadiatorCUDA::computeBinning()",
       MPI_Allreduce(&alphaMinTotLocal, &alphaMinTotGlobal, 1, 
		     MPIStructDef::getMPIType(&alphaMinTotLocal), 
		     MPI_MIN, PE::GetPE().GetCommunicator(m_namespace)));
    MPIError::getInstance().check
      ("MPI_Allreduce", "ParadeRadiatorCUDA::computeBinning()",
       MPI_Allreduce(&alphaMaxTotLocal, &alphaMaxTotGlobal, 1, 
		     MPIStructDef::getMPIType(&alphaMaxTotLocal), 
		     MPI_MAX, PE::GetPE().GetCommunicator(m_namespace)));
    
    alphamin_tot = alphaMinTotGlobal;
    alphamax_tot = alphaMaxTotGlobal;
  }
  
  vctBins[0] = alphamin_tot;
  vctBins[nbBins-1] = alphamax_tot;
  
  CFLog(VERBOSE,"ParadeLibrary::computeBinning () => [alphamin_tot, alphamax_tot] = [" 
	<< alphamin_tot << ", "  << alphamax_tot <<"\n");
  
  computeAveragedBins(nbBins,2, vctBins);
  
  CFLog(INFO, "ParadeRadiatorCUDA::computeBinning() took " << timer.elapsed() << "s\n"); 
  CFLog(VERBOSE, "ParadeRadiatorCUDA::computeBinning() => END\n");
}
    
//////////////////////////////////////////////////////////////////////////////
  
void ParadeRadiatorCUDA::computeBanding() 
{  
  CFLog(VERBOSE, "ParadeRadiatorCUDA::computeBanding() => START\n");

  Stopwatch<WallTime> stp;
  stp.start();
  
  const CFuint totalNbCells = m_pstates->getSize();
  const CFuint nbCols = m_nbPoints*3;
  CFuint nbCells = m_data.size()/nbCols;
  cf_assert((!m_saveMemory && nbCells == totalNbCells) || 
	    ( m_saveMemory && nbCells <= totalNbCells));
  
  if (m_saveMemory) {
    CFuint totalNbCells = 0;
    MPIError::getInstance().check
      ("MPI_Allreduce", "ParadeRadiatorCUDA::computeBinning()",
       MPI_Allreduce(&nbCells, &totalNbCells, 1, MPIStructDef::getMPIType(&nbCells), 
		     MPI_SUM, PE::GetPE().GetCommunicator(m_namespace)));
    cf_assert(totalNbCells == m_pstates->getSize());
  }

  // Banding
  CFreal alphamin_tot = m_data[0];
  CFreal alphamax_tot = m_data[0];
  
  for(CFuint i=0;i<m_nbPoints;++i) {
    for(CFuint s=0;s<nbCells;s++) {
      const CFreal wavelength = m_data[s*nbCols + i*3];
      if(wavelength < alphamin_tot) {
	alphamin_tot = wavelength;
      }
      if(wavelength > alphamax_tot) {
	alphamax_tot = wavelength;
      }
    }
  }
  
  CFreal alphamax_totGlobal = 0.;
  CFreal alphamin_totGlobal = 1e10;
  
  MPIError::getInstance().check
    ("MPI_Allreduce", "ParadeRadiatorCUDA::computeBanding()",
     MPI_Allreduce(&alphamin_tot, &alphamin_totGlobal, 1, 
		   MPIStructDef::getMPIType(&alphamin_tot), 
		   MPI_MIN, PE::GetPE().GetCommunicator(m_namespace)));
  
  MPIError::getInstance().check
    ("MPI_Allreduce", "ParadeRadiatorCUDA::computeBanding()",
     MPI_Allreduce(&alphamax_tot, &alphamax_totGlobal, 1, 
		   MPIStructDef::getMPIType(&alphamax_tot), 
		   MPI_MAX, PE::GetPE().GetCommunicator(m_namespace)));
  
  const CFreal alpha_minlog = std::log(alphamin_totGlobal);
  const CFreal alpha_maxlog = std::log(alphamax_totGlobal);

  CFLog(INFO,"ParadeLibrary::computeBanding () => [alpha_minlog, alpha_maxlog] = ["
	<< alpha_minlog << ", " << alpha_maxlog << "]\n");
  
  const CFreal dy = (alpha_maxlog-alpha_minlog) / (m_nbBands-1);
  
  CFLog(INFO,"ParadeLibrary::computeBanding() => dy = " << dy <<"\n");
  
  LocalArray<CFreal>::TYPE vctBins(0., m_nbBands);
  
  vctBins = 0.;
  for(int i = 0; i<m_nbBands; ++i) {
    vctBins[i] = std::exp(alpha_minlog + (dy * i));
    CFLog(VERBOSE,"ParadeLibrary::computeBanding() => vctBins(" << i << ") = " << vctBins[i] <<"\n");
  }
  
  // AL: I stop here 
  //To search for minimum alpha and maximum
  computeAveragedBins(m_nbBands, 0, vctBins);
  
  CFLog(INFO, "ParadeRadiatorCUDA::computeBanding() took " << stp.read() << "s\n");
  CFLog(VERBOSE, "ParadeRadiatorCUDA::computeBanding() => END\n");
}

//////////////////////////////////////////////////////////////////////////////

void ParadeRadiatorCUDA::computeBinningBanding() 
{ 
  CFLog(VERBOSE, "ParadeRadiatorCUDA::computeBinningBanding() => START\n");
  Stopwatch<WallTime> stp;
  stp.start();
  CFLog(INFO, "ParadeRadiatorCUDA::computeBinningBanding() took "<< stp.read() << "s\n");  
  CFLog(VERBOSE, "ParadeRadiatorCUDA::computeBinningBanding() => END\n");
}
    
//////////////////////////////////////////////////////////////////////////////


  
//////////////////////////////////////////////////////////////////////////////

void ParadeRadiatorCUDA::computeAveragedBins(const CFuint nbBinsre, 
					     const CFuint testID,
					     LocalArray<CFreal>::TYPE& vctBins)
{
  const CFuint totalNbCells = m_pstates->getSize();
  const CFuint nbCols = m_nbPoints*3;
  CFuint nbCells = m_data.size()/nbCols;
  
  //alpha_avbin is average value for absorptivity for each bin
  SafePtr<SocketBundle> sockets = m_radPhysicsHandlerPtr->getDataSockets();
  DataHandle<CFreal> alpha_avbin = sockets->alpha_avbin; // array w GLOBAL cell size
  DataHandle<CFreal> B_bin = sockets->B_bin;             // array w GLOBAL cell size
  
  if (alpha_avbin.size() != nbBinsre*totalNbCells) {
    CFLog(ERROR, "ParadeRadiatorCUDA::computeAveragedBins() => alpha_avbin.size() != nbBinsre*totalNbCells => " << alpha_avbin.size() << " != " <<  nbBinsre*totalNbCells << "\n");
    cf_assert(alpha_avbin.size() == nbBinsre*totalNbCells);
  }
  if (B_bin.size() != nbBinsre*totalNbCells) {
    CFLog(ERROR, "ParadeRadiatorCUDA::computeAveragedBins() => B_bin.size() != nbBinsre*totalNbCells => " << B_bin.size() << " != " <<  nbBinsre*totalNbCells << "\n");
    cf_assert(B_bin.size() == nbBinsre*totalNbCells);
  }
  
  // from now only local arrays (=global if m_saveMemory==false) are used
  m_alpha_bin.resize(nbBinsre*nbCells);     // array w LOCAL cell size
  m_emission_bin.resize(nbBinsre*nbCells);  // array w LOCAL cell size
  
  cf_assert((nbCells < totalNbCells && m_nbProc > 1) || 
	    (nbCells == totalNbCells && m_nbProc == 1));
  
  LocalArray<CFreal>::TYPE alpha_avbinCurr(0., nbBinsre*nbCells);
  LocalArray<CFreal>::TYPE B_binCurr(0., nbBinsre*nbCells);
  
  const CFuint threads = CudaEnv::CudaDeviceManager::getInstance().getNThreads();
  const CFuint blocks = 
    CudaEnv::CudaDeviceManager::getInstance().getBlocksPerGrid(nbCells*m_nbPoints);
  
  CFLog(INFO, "ParadeRadiatorCUDA::computeAveragedBins() => [blocks, threads] = [" 
	<< blocks << ", " << threads << "]\n");
  
  // copy input data from CPU to GPU
  vctBins.put(); 
  m_data.put();

  const CFuint isEquil = (const CFuint)m_Equilibrium;
  // call a kernel that computes bins 
  computeCellBinsKernel<<<blocks, threads>>>
    (isEquil, m_nbPoints, nbCells, nbBinsre, testID, m_dWav, vctBins.ptrDev(), m_data.ptrDev(),
     m_alpha_bin.ptrDev(), m_emission_bin.ptrDev(), B_binCurr.ptrDev());
  
  // copy output data from GPU to CPU
  m_alpha_bin.get();
  m_emission_bin.get();
  B_binCurr.get();
  
  /*for(CFuint k=1;k<nbBinsre;++k) {
    for(CFuint j=0;j<nbCells;++j) {
    CFLog(DEBUG_MAX,"ParadeLibrary::computeproperties () => m_alpha_bin(" << k << "," << j << ") = " << m_alpha_bin[nbBinsre*j+k] <<"\n");
    CFLog(DEBUG_MAX,"ParadeLibrary::computeproperties () => B_binCurr(" << k << "," << j << ") = " << B_binCurr[nbBinsre*j+k] <<"\n");
    }
    }*/
  
  for(CFuint j=0;j<nbCells;++j) {
    for(CFuint k=1;k<nbBinsre;++k) {
      const CFuint idx0 = nbBinsre*j;
      alpha_avbinCurr[idx0] = 0.;
      const CFuint idx = k + idx0;
      // AL: is this fix needed to mask an error or it is supposed to be like this?
      if(B_binCurr[idx] != 0.) {
	alpha_avbinCurr[idx] = m_alpha_bin[idx] / B_binCurr[idx];
	CFLog(DEBUG_MED,"ParadeRadiatorCUDA::computeAverageBins() => alpha_avbinCurr(" << k << "," << j << ") = "<< alpha_avbinCurr[idx] <<"\n");
      }
      else {
	alpha_avbinCurr[idx] = 0.;
	CFLog(DEBUG_MED,"ParadeRadiatorCUDA::computeAverageBins() => alpha_avbinCurr(" << k << ","<< j << ") = "<< alpha_avbinCurr[idx] <<"\n");
      }
    }
  }
  
  if (m_saveMemory) {
    // here we need to gather all the entries for alpha_avbin and B_bin from all processes, so that
    // every process keeps a global storage of them
    CFuint minSizeToSend = 0;
    CFuint maxSizeToSend = 0;
    vector<int> recvCounts(m_nbProc, 0);
    vector<int> displs(m_nbProc, 0);
    computeRecvCountsDispls(totalNbCells, nbBinsre, minSizeToSend, maxSizeToSend, recvCounts, displs);
    const CFuint sendSize = nbCells*nbBinsre;
    cf_assert(sendSize <= maxSizeToSend);
    cf_assert(sendSize >= minSizeToSend);
    cf_assert(sendSize <= maxSizeToSend);
    cf_assert(sendSize >= minSizeToSend);
    
    MPIError::getInstance().check
      ("MPI_Allgatherv", "ParadeRadiatorCUDA::computeAverageBins() => alpha_avbin",
       MPI_Allgatherv(&alpha_avbinCurr[0], sendSize, MPIStructDef::getMPIType(&alpha_avbinCurr[0]),
		      &alpha_avbin[0], &recvCounts[0], &displs[0],  MPIStructDef::getMPIType(&alpha_avbin[0]),
		      PE::GetPE().GetCommunicator(m_namespace)));
    
    MPIError::getInstance().check
      ("MPI_Allgatherv", "ParadeRadiatorCUDA::computeAverageBins() => B_bin",
       MPI_Allgatherv(&B_binCurr[0], sendSize, MPIStructDef::getMPIType(&B_binCurr[0]),
		      &B_bin[0], &recvCounts[0], &displs[0],  MPIStructDef::getMPIType(&B_bin[0]),
		      PE::GetPE().GetCommunicator(m_namespace)));
  }
  
  // To be commented out after verification
  if(PE::GetPE().GetRank(m_namespace) == 0) {
   
    /*for(CFuint j=0;j<totalNbCells;++j) {
	for(CFuint k=1;k<nbBinsre;++k) {
	CFLog(DEBUG_MAX,"alpha (" << k << "," << j << ") = " << alpha_avbin[k + nbBinsre*j] << "\n");
	}
	}*/
    ofstream fout1("alpha.txt");
    for(CFuint j=0;j<totalNbCells;++j) {
      for(CFuint k=1;k<nbBinsre;++k) {
	fout1 << "alpha (" << k << "," << j << ") = " << 
	  alpha_avbin[k + nbBinsre*j] << "\n";
      }
    }
    fout1.close();
    
    ofstream fout2("beta.txt");
    for(CFuint j=0;j<totalNbCells;++j) {
      for(CFuint k=1;k<nbBinsre;++k) {
	fout2 << "beta (" << k << "," << j << ") = " << 
	  B_bin[k + nbBinsre*j] << "\n";
      }
    }
    fout2.close();
  }
  
  PE::GetPE().setBarrier(m_namespace);
}

//////////////////////////////////////////////////////////////////////////////

} // namespace RadiativeTransfer

} // namespace COOLFluiD

